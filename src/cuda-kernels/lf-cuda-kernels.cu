#include "hip/hip_runtime.h"
/**
 * LF-TADCG-CUDA-DISP - Code for estimating the disparity map of a light field
 *
 * Copyright (C) 2017 Adam Stacey
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "lf-cuda-kernels.h"
#include "../lf-depth-est.h"

dim3 cuda_threads_per_block(16, 16);
dim3 cuda_num_blocks(32, 32); //temporary hard code for 512 * 512 images

/**
 * Arrange CUDA blocks to accomodate image.  Can be optimised further.
 * @param image_cols
 * @param image_rows
 * @param threads
 */
void setCudaParamsForImage(size_t image_cols, size_t image_rows, dim3 threads) {
  cuda_threads_per_block.x = threads.x;
  cuda_threads_per_block.y = threads.y;
  cuda_threads_per_block.z = threads.z;
  cuda_num_blocks = dim3(
      static_cast<int>(std::ceil(image_cols /
          static_cast<double>(cuda_threads_per_block.x))),
      static_cast<int>(std::ceil(image_rows /
          static_cast<double>(cuda_threads_per_block.y))));
}

void setCudaParamsForImage(cv::Mat image, dim3 threads) {
  setCudaParamsForImage(image.cols, image.rows, threads);
}

void setCudaParamsForImage(cv::cuda::GpuMat image, dim3 threads) {
  setCudaParamsForImage(image.cols, image.rows, threads);
}

/**
 * Populate a gpuMat for a u,v shift when computing the 4D shear of the light field
 * @param x_map_gpu
 * @param y_map_gpu
 * @param v_shift
 * @param u_shift
 */
__global__ void ShiftMapCalcKernel(cv::cuda::PtrStepSzf x_map_gpu,
                                   cv::cuda::PtrStepSzf y_map_gpu,
                                   float v_shift,
                                   float u_shift) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < x_map_gpu.cols && y < x_map_gpu.rows && y >= 0 && x >= 0) {
    x_map_gpu(y, x) = x + v_shift;
    y_map_gpu(y, x) = y + u_shift;
  }
}

void ShiftMapCalc(cv::cuda::PtrStepSzf x_map_gpu,
                  cv::cuda::PtrStepSzf y_map_gpu,
                  float v_shift,
                  float u_shift,
                  hipStream_t stream) {
  ShiftMapCalcKernel << < cuda_num_blocks, cuda_threads_per_block, 0, stream >> > (x_map_gpu, y_map_gpu,
      v_shift, u_shift);
}


/**
 * Calculate the TAD C+G contribution for a sheared light field (colour)
 * @param t_cg_m
 * @param ref
 * @param ref_grad
 * @param defoc
 * @param dx
 * @param dy
 */
__global__ void ContribToTadCGMeanColourKernel(cv::cuda::PtrStepSzf t_cg_m,
                                               cv::cuda::PtrStepSzf ref,
                                               cv::cuda::PtrStepSzf ref_grad,
                                               cv::cuda::PtrStepSzf defoc,
                                               cv::cuda::PtrStepSzf dx,
                                               cv::cuda::PtrStepSzf dy) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  float temp_grad0, temp_grad1, temp_grad2;
  if (x < t_cg_m.cols && y < t_cg_m.rows && y >= 0 && x >= 0) {

    temp_grad0 = fminf(fabsf(ref_grad(y, (3 * x)) - (0.5f * dx(y, (3 * x)) + 0.5f * dy(y, (3 * x)))), TAD_G_TAO);
    temp_grad1 =
        fminf(fabsf(ref_grad(y, (3 * x) + 1) - (0.5f * dx(y, (3 * x) + 1) + 0.5f * dy(y, (3 * x) + 1))), TAD_G_TAO);
    temp_grad2 =
        fminf(fabsf(ref_grad(y, (3 * x) + 2) - (0.5f * dx(y, (3 * x) + 2) + 0.5f * dy(y, (3 * x) + 2))), TAD_G_TAO);
    t_cg_m(y, (3 * x)) += (TAD_CG_ALPHA * fminf(fabsf(ref(y, (3 * x)) - defoc(y, (3 * x))), TAD_C_TAO)) +
        ((1 - TAD_CG_ALPHA) * temp_grad0);
    t_cg_m(y, (3 * x) + 1) += (TAD_CG_ALPHA * fminf(fabsf(ref(y, (3 * x) + 1) - defoc(y, (3 * x) + 1)), TAD_C_TAO)) +
        ((1 - TAD_CG_ALPHA) * temp_grad1);
    t_cg_m(y, (3 * x) + 2) += (TAD_CG_ALPHA * fminf(fabsf(ref(y, (3 * x) + 2) - defoc(y, (3 * x) + 2)), TAD_C_TAO)) +
        ((1 - TAD_CG_ALPHA) * temp_grad2);
  }
}

void ContribToTadCGMeanColour(cv::cuda::PtrStepSzf t_cg_m,
                              cv::cuda::PtrStepSzf ref,
                              cv::cuda::PtrStepSzf ref_grad,
                              cv::cuda::PtrStepSzf defoc,
                              cv::cuda::PtrStepSzf dx,
                              cv::cuda::PtrStepSzf dy, hipStream_t stream) {
  ContribToTadCGMeanColourKernel << < cuda_num_blocks, cuda_threads_per_block, 0, stream >> > (t_cg_m, ref,
      ref_grad, defoc, dx, dy);
}

/**
 * Calculate the TAD C+G contribution for a sheared light field (grayscale)
 * @param t_cg_m
 * @param ref
 * @param ref_grad
 * @param defoc
 * @param dx
 * @param dy
 */
__global__ void ContribToTadCGMeanBWKernel(cv::cuda::PtrStepSzf t_cg_m,
                                           cv::cuda::PtrStepSzf ref,
                                           cv::cuda::PtrStepSzf ref_grad,
                                           cv::cuda::PtrStepSzf defoc,
                                           cv::cuda::PtrStepSzf dx,
                                           cv::cuda::PtrStepSzf dy) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  float temp_grad;
  if (x < t_cg_m.cols && y < t_cg_m.rows && y >= 0 && x >= 0) {

    temp_grad = fminf(fabsf(ref_grad(y, (3 * x)) - (0.5f * dx(y, x) + 0.5f * dy(y, x))), TAD_G_TAO);
    t_cg_m(y, x) += (TAD_CG_ALPHA * fminf(fabsf(ref(y, x) - defoc(y, x)), TAD_C_TAO)) +
        ((1 - TAD_CG_ALPHA) * temp_grad);
  }
}

void ContribToTadCGMeanBW(cv::cuda::PtrStepSzf t_cg_m,
                          cv::cuda::PtrStepSzf ref,
                          cv::cuda::PtrStepSzf ref_grad,
                          cv::cuda::PtrStepSzf defoc,
                          cv::cuda::PtrStepSzf dx,
                          cv::cuda::PtrStepSzf dy, hipStream_t stream) {
  ContribToTadCGMeanBWKernel << < cuda_num_blocks, cuda_threads_per_block, 0, stream >> > (t_cg_m, ref, ref_grad,
      defoc, dx, dy);
}

__global__ void AddColourKernel(cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<float> b) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < a.cols && y < a.rows && y >= 0 && x >= 0) {
    a(y, (3 * x) + 0) += b(y, (3 * x) + 0);
    a(y, (3 * x) + 1) += b(y, (3 * x) + 1);
    a(y, (3 * x) + 2) += b(y, (3 * x) + 2);
  }
}

void AddColour(cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<float> b) {
  AddColourKernel << < cuda_num_blocks, cuda_threads_per_block >> > (a, b);
}

__global__ void AbsdiffColourKernel(cv::cuda::PtrStepSz<float> a,
                                    cv::cuda::PtrStepSz<float> b,
                                    cv::cuda::PtrStepSz<float> output) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < a.cols && y < a.rows && y >= 0 && x >= 0) {
    output(y, (3 * x) + 0) = fabsf(a(y, (3 * x) + 0) - b(y, (3 * x) + 0));
    output(y, (3 * x) + 1) = fabsf(a(y, (3 * x) + 1) - b(y, (3 * x) + 1));
    output(y, (3 * x) + 2) = fabsf(a(y, (3 * x) + 2) - b(y, (3 * x) + 2));
  }
}

void AbsdiffColour(cv::cuda::PtrStepSz<float> a, cv::cuda::PtrStepSz<float> b,
                   cv::cuda::PtrStepSzf output, hipStream_t stream) {
  AbsdiffColourKernel << < cuda_num_blocks, cuda_threads_per_block, 0, stream >> > (a, b, output);
}
